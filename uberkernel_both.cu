#include "hip/hip_runtime.h"
// -*- c++ -*-


//*** ***** FORCES EVALUATION KERNELS ***** ***//
//***********************************************



__global__ void Calc_Both( float4 *pos, short *AtomType_d, float *Potentials_d, float3 *forces )
{
  
  __shared__ float4 spos[BLOCKSIZE];           //shared positions
  //__shared__ float  spots[BLOCKSIZE];
  __shared__ short  types[BLOCKSIZE];          //shared atomic types
  
  float  myfor[3], r[3], dist; // 7 floats
  float  mypot = 0.0f;         // 8
  float4 mypos;                // 12
  short  mytype;                 
  
  //get the right block index (separation between force and energy)
  int blockx = blockIdx.x;
  int isE = 0;
  if(blockIdx.x >= gridDim.x/2){
    blockx -= gridDim.x/2;
    isE = 1;
  }

  //get the image index
  int imgidx = (blockx/BperIMG_d); //index of my image in shared to save space
  blockx -= imgidx*BperIMG_d;

  int idx = (blockx * BLOCKSIZE + threadIdx.x); //index of the atom this thread will take care of
  short isok = (idx < NAtoms_d);
  idx *= isok;

  int windex = idx + imgidx*NAtoms_d;  //16 floats used in registers!!!


  mypos  = pos[windex];       //position of the atom for which we will calculate the potential contribution
  mytype = AtomType_d[idx];//this atom type
  myfor[0] = 0.0f;
  myfor[1] = 0.0f;
  myfor[2] = 0.0f;  //set the force to 0


  //forces[threadIdx.x+blockIdx.x*BLOCKSIZE] = make_float3(blockIdx.x*1000.0f+threadIdx.x,blockx, imgidx);

  //return;

  //loop on all groups of consecutive BLOCKSIZE particles (will loop AAALL atoms!!!)
  for(int i=0; i < BperIMG_d; i++) //loops over the Blocks needed for one image of the system (NAtoms!)
    {

      //every thread loads one position/charge & types from the global memory
      if(threadIdx.x + i*BLOCKSIZE < NAtoms_d)
	{
	  spos[threadIdx.x] = pos[threadIdx.x + i*BLOCKSIZE+NAtoms_d*imgidx];
	  types[threadIdx.x] = AtomType_d[threadIdx.x + i*BLOCKSIZE];
	}
      __syncthreads();
      //*************************************************************
      
      //now loop on the loaded elements and sum the contributions
      for(int j=0; j<BLOCKSIZE; j++)
	{

	  //stop summing if the index of j is already out
	  if(i*BLOCKSIZE + j >= NAtoms_d)
	    break;
	  
	  //exclude selfterms
	  if( j+i*BLOCKSIZE != idx )
	    {
	      
	      //compute the distance
	      r[0] = spos[j].x - mypos.x;
	      r[1] = spos[j].y - mypos.y;
	      r[2] = spos[j].z - mypos.z;
	      dist = r[0]*r[0] + r[1]*r[1] + r[2]*r[2];
	      dist = sqrtf(dist);
	      
	      
	      if(isE)
		{
		  mypot +=  0.5f * spos[j].w*mypos.w * 14.39964524f / dist;
		  mypot +=  0.5f * PairPotential( dist, mytype, types[j] );	    
		}
	      else
		{
		  //get the coulomb part
		  CoulombForce(r, dist, spos[j].w*mypos.w, myfor );
		  //pair potential part
		  PairForce(r, dist, mytype, types[j], myfor);
		}
	      
	    }
	}
      //-------------------------------------------------------------


      __syncthreads();
      
    }

  //save the results in the global memory
  if(isE)
    {
      spos[threadIdx.x].x = mypot * isok; //every thread writes the partial in the shared
      __syncthreads();

      //the first thread of each block copies the result in the global memory
      mypot = 0.0f;
      if(threadIdx.x == 0)
	{
	  for(int i=0;i<BLOCKSIZE;i++)
	    {
	      mypot += spos[i].x;
	    }
	  Potentials_d[blockx+BperIMG_d*imgidx] = mypot;
	}
      
    }
  else
    {
      //__syncthreads();
      if(isok == 1){
	forces[windex] = make_float3(myfor[0],myfor[1],myfor[2]);
	//forces[windex] = make_float3(blockIdx.x*100.0f+threadIdx.x, blockx,windex);
      }
    }
  
  

}




int GetForcesEnergy()
  {
    dim3 dimBlock_f(BLOCKSIZE);
    dim3 dimGrid_f(2 * Run.BperIMG * Run.MDimages);
    if(dimGrid_f.x == 0) dimGrid_f = 2;
    
    int i,img;
    float energy;

    for(i=0;i<Run.NAtoms;i++)
      Potentials_h[i] = 0.0f;
    
    //printf("atoms=%i gridsize would be %i\n",Run.NAtoms, dimGrid_f.x );

    
    Calc_Both<<<dimGrid_f, dimBlock_f>>>(Charges_d, AtomType_d, Potentials_d, forces_d);
    hipDeviceSynchronize();
    
    hipMemcpy(forces_h, forces_d, f3_NObj, hipMemcpyDeviceToHost);           //copy back the forces
    hipMemcpy(Potentials_h, Potentials_d, f1_NObj, hipMemcpyDeviceToHost);

    
    //    for(i=0; i<Run.MDimages*Run.BperIMG; i++)
    //      printf("pot %i = %f\n",i,Potentials_h[i]);

    //sum up the potentials given by each block
    for(img=0; img<Run.MDimages; img++)
      {
	energy = 0.0f;
	for(i=0;i<Run.BperIMG;i++)
	  energy += Potentials_h[i+img*Run.BperIMG];
	Etot[img] = energy;
	//printf("Energy[%i] is %f\n",img,energy);
      }
   
    /*
    FILE *fp = fopen("force2.out","w");
    
    //for(img=0;img<Run.MDimages;img++)
      for(i=0;i<2*Run.MDimages*Run.NAtoms;i++)
	{
	  //fprintf(fp,"%8.5f %8.5f %8.5f - %8.5f \n",Charges_h[i].x,Charges_h[i].y,Charges_h[i].z,Charges_h[i].w);
	  //fprintf(fp,"%8.5f %8.5f %8.5f \n",forces_h[i].x,forces_h[i].y,forces_h[i].z);
	  //fprintf(fp,"atom%i %f %f %f ...  %f\n",i,forces_h[i].x-forces_h[i+Run.NAtoms].x, forces_h[i].y-forces_h[i+Run.NAtoms].y, 
	  //	forces_h[i].z-forces_h[i+Run.NAtoms].z,Charges_h[i].w);
	  //fprintf(fp,"index%i blk%f  thr%f  aidx%f  isok %f\n",i,forces_h[i].x, forces_h[i].y,forces_h[i].z,
	  //	  Potentials_h[i]);
	  fprintf(fp,"index%i %f  %f %f %f\n",i,Potentials_h[i],Charges_h[i].x,Charges_h[i].y,Charges_h[i].z);
	}
	fclose(fp);*/

    
    return true;

  }
