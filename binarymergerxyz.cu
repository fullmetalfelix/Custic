// -*- c++ -*-


#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <stdlib.h>
//#include <strings.h>
#define true 1
#define false 0


//take nloops for one image and print them to a single binary
//file for Cinema 4D loader script

int StartsWith(char strIn[], char strRef[]);



int main (int argc, char* argv[])
{

  char filename[100], nameout[100],aname[5];
  int i,j,k,nloops,img, natoms, nimages;
  int frames;
  FILE *fin, *fout;
  short3 pos;
  float3 speed;

  if(argc<4){
    printf("Insufficient parameters! Specify the base name, the amount of loops and the image number\n");
    return 0;
  }
  
  
  nloops = atoi(argv[2]);
  img  = atoi(argv[3]);
  frames = 0;

  sprintf(nameout , "%s.trajectory.img%i.xyz",argv[1],img); //open the output file

  //open the first file and get NATM and NIMG
  //all files should have the same amount of atoms, images and frames

  sprintf(filename,"%s.loop%i.ctj",argv[1],1);
  printf("Opening %s\n",filename);
  fin = fopen(filename,"rb");
  fout= fopen(nameout ,"w");
  fread(&natoms, sizeof(int),1,fin);
  fread(&nimages,sizeof(int),1,fin);

  printf("NUMBER OF ATOMS: %i\n",natoms);
  
  //this counts the frames
  while(!feof(fin)){ //loop until EOF
    frames++;
    for(i=0;i<natoms;i++){  //loop on all atoms

      if((int)fread(aname,sizeof(char),5,fin) == 0)   //read the first atom
	break;

      fseek(fin,18*(img-1), SEEK_CUR);    //skip (img-1)(3short+3float)
      fread(&pos,sizeof(short3),1,fin);   //->position as float3
      fread(&speed,sizeof(float3),1,fin); //->speed as float3
      //printf("--%s-- %i %i %i -- %f %f %f\n",aname,pos.x,pos.y,pos.z,speed.x,speed.y,speed.z);
  
      fseek(fin,18*(nimages-img), SEEK_CUR);  //now skip (NIMG-img)(3short+3float)
    }
  }
  printf("TOTAL FRAMES IN FILE: %i\n",frames);
  fclose(fin);
  
  fprintf(fout,"%i\n\n",natoms);

  for(i=1;i<=nloops;i++){   //loop over the files

    sprintf(filename,"%s.loop%i.ctj",argv[1],i); //make the filename
    fin = fopen(filename,"rb");  //open it
    fseek(fin,8,SEEK_CUR);  //skip the first 2 int      
    
    for(k=0;k<frames-1;k++){   //loop over all frames in the current file
      for(j=0;j<natoms;j++){   //loop over all atoms
	if((int)fread(aname,sizeof(char),5,fin) == 0)
	  break;
	fseek(fin,18*(img-1), SEEK_CUR);  //skip non wanted images
	//read data for the wanted image
	fread(&pos,sizeof(short3),1,fin);   //->position as float3
	fread(&speed,sizeof(float3),1,fin); //->speed as float3
	fseek(fin,18*(nimages-img), SEEK_CUR);  //skip the subsequent images
	
	speed.x = (float)pos.x/100.0f;
	speed.y = (float)pos.y/100.0f;
	speed.z = (float)pos.z/100.0f;

	fprintf(fout,"%s %f %f %f\n",aname,speed.x,speed.y,speed.z);
      }
      fprintf(fout,"\n\n");
    }
    fclose(fin);
  }

  fclose(fout);

}





int StartsWith(char strIn[], char strRef[])
{
  int i;
  
  for(i=0;i<strlen(strRef);i++)
    {
      if(strIn[i]!=strRef[i])
	return false;
      //printf("%c %c\n",strIn[i],strRef[i]);
    }
  return true;
}



